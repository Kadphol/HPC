#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

const int THREADS = 128; //threads per block
const int trial_number = 1024; //trial numbers per thread
const int BLOCKS = 16; //blocks per grid
const float PI = 3.1415926535;

__global__ void pi_estimation(float *pi, hiprandState *states)
{
    unsigned int threadID = threadIdx.x + blockDim.x * blockIdx.x;
    int count = 0;
    float x,y,z;

    hiprand_init(0, threadID, 1, &states[threadID]); //(seed, sequence number, offset, curandState)

    for(int i = 0; i < trial_number; i++)
    {
        x = hiprand_uniform(&states[threadID]);//return sequence number of pseudorandom uniformly distributed.
        y = hiprand_uniform(&states[threadID]);
        z = (x*x + y*y);
        if(z <= 1.0f) //if x,y in unit circle
        {
            count += 1;
        }
    }
    pi[threadID] = 4.0f * count/(float)trial_number; //estimate PI value 4*count/number of trial
} 

int main(int argc, char *argv[])
{
    float host[THREADS * BLOCKS];
    float *device;
    hiprandState *deviceStates;

    hipMalloc((void **)&device, THREADS * BLOCKS * sizeof(float));
    hipMalloc((void **)&deviceStates, THREADS * BLOCKS * sizeof(hiprandState));

    pi_estimation<<<BLOCKS,THREADS>>>(device,deviceStates); //call kernel
    hipMemcpy(host, device, THREADS * BLOCKS * sizeof(float), hipMemcpyDeviceToHost); //copy estimated pi value from device to host
    float pi = 0.0;
    for(int i = 0; i < THREADS * BLOCKS; i++)
    {
        pi += host[i];
    }
    pi /= (THREADS * BLOCKS); //get average of each PI value from each thread.
    printf("Monte Carlo PI estimation %d times\n", THREADS * BLOCKS * trial_number);
    printf("PI estimation: %.10f\n",pi);
    printf("Error: %.10\n\n",pi-PI);
    hipFree(device);
    hipFree(deviceStates);
    return 0;
}
